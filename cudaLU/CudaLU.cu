#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <omp.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>  
#include "hip/hip_runtime.h"
#include ""

using namespace std;
#define TILE 3

//Get input at the argument in the properties
bool inputMatrix(int argc, char* argv[], int& n, int& isPrint)
{
	bool correct = true;

	if (argc < 2)
	{
		cout << "Arguments:X Y" << endl;
		cout << "X : Matrix size [N x N]" << endl;
		cout << "Y = 1: print the input/output matrix if X < 10" << endl;
		correct = false;
	}
	else
	{
		//get matrix size in argument pos 1
		n = atoi(argv[1]);
		if (n <= 0)
		{
			cout << "Matrix size must be larger than 0" << endl;
			correct = false;
		}

		//is print the input/output matrix
		if (argc >= 3)
		{
			isPrint = (atoi(argv[2]) == 1 && n <= 9) ? 1 : 0;
		}
		else
		{
			isPrint = 0;
		}
	}
	return correct;
}

//Initialize the value of matrix a[n x n]
void InitializeMatrix(float**& a, int n)
{
	a = new float* [n];
	a[0] = new float[n * n];

	for (int i = 1; i < n; i++)
	{
		a[i] = a[i - 1] + n;
	}

	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			if (i == j)
			{
				a[i][j] = (((float)i + 1) * ((float)i + 1)) / (float)2;
			}

			else
			{
				a[i][j] = (((float)i + 1) + ((float)j + 1)) / (float)2;
			}
		}
	}
}

//Initialize the value of matrix L[n x n] for lower triangular matrix
void InitializeLowerMatrix(float**& L, int n) {
	L = new float* [n];
	L[0] = new float[n * n];

	for (int i = 1; i < n; i++)
	{
		L[i] = L[i - 1] + n;
	}

	for (int j = 0; j < n; j++)
	{
		for (int i = 0; i < n; i++)
		{
			if (i == j)
			{
				L[j][i] = 1;
			}
			else
			{
				L[j][i] = 0;
			}
		}
	}
}

//Compute the LU Decomposition for matrix a[n x n] and L[n x n]
__global__ void ComputeLUDecomposition(float* a, float* L, int n)
{
	//extern __shared__ float pivot;
	//Define the variables
	float pivot, gmax, pmax, temp;
	int  pindmax, gindmax, i, j, k;

	int tx = threadIdx.x; //threadIdx.x to access thread index within block
	int ty = threadIdx.y;
	int Row = blockIdx.x * TILE + tx; // blockIdx.x to access block index within grid
	int Col = blockIdx.y * TILE + ty;

	// Synchronize to make sure the sub-matrices are loaded
	// before starting the computation
	__syncthreads(); //a barrier use to prevent data hazards
	if (Row < n && Col < n)
	{
		//Perform rowwise elimination
		for (k = 0; k < n - 1; k++)
		{
			gmax = 0.0;

			//Find the pivot row among rows k, k+1,...n
			//Each thread works on a number of rows to find the local max value pmax
			//Then update this max local value to the global variable gmax
			{
				pmax = 0.0;
				for (i = k; i < n; i++)
				{
					temp = abs(a[i * n + k]);

					if (temp > pmax)
					{
						pmax = temp;
						pindmax = i;
					}
				}

				if (gmax < pmax)
				{
					gmax = pmax;
					gindmax = pindmax;
				}
			}
			//If matrix is singular set the flag & quit
			if (gmax == 0)
			{
				return;
			}

			//Swap rows if necessary
			if (gindmax == k)
			{
				for (j = k; j < n; j++)
				{
					temp = a[gindmax * n + j];
					a[gindmax * n + j] = a[k * n + j];
					a[k * n + j] = temp;
				}
			}

			//Compute the pivot
			pivot = -1.0 / a[k * n + k];

			//Perform row reductions
			for (i = k + 1; i < n; i++)
			{
				temp = pivot * a[i * n + k];
				L[i * n + k] = ((-1.0) * temp);
				for (j = k; j < n; j++)
				{
					a[i * n + j] = a[i * n + j] + temp * a[k * n + j];
				}
			}
		}
	}
	return;
}

//Print lower triangular matrix	
void PrintMatrix(float** a, int n)
{
	for (int i = 0; i < n; i++)
	{
		cout << "Row " << (i + 1) << ":\t";
		for (int j = 0; j < n; j++)
		{
			printf("%.2f\t", a[i][j]);
		}
		cout << endl;
	}
}

//Print upper triangular matrix	
void PrintMatrixU(float** a, int n)
{
	for (int i = 0; i < n; i++)
	{
		cout << "Row " << (i + 1) << ":\t";
		for (int j = 0; j < n; j++)
		{
			if (j < i) {
				a[i][j] = 0;
			}
			printf("%.2f\t", a[i][j]);
		}
		cout << endl;
	}
}

int main(int argc, char* argv[])
{
	int n = 0, isPrintMatrix = 0;
	float** a;
	float** L;
	float* da, * dl, * du; //device pointers
	double runtime;
	bool correct;

	if (inputMatrix(argc, argv, n, isPrintMatrix) == false)
	{
		return 1;
	}

	cout << "Cuda 1 - gpu matrix " << endl;
	cout << "matrix size is " << n << endl;

	runtime = clock() / (double)CLOCKS_PER_SEC;

	//Initialize the value of matrix A[n x n]
	InitializeMatrix(a, n);
	InitializeLowerMatrix(L, n);

	if (isPrintMatrix == 1)
	{
		cout << "CUDA LU decomposition" << endl;
		cout << "=========================" << endl << endl;
		cout << "Generated : " << n << " x " << n << " Matrix" << endl;
		PrintMatrix(a, n);
	}

	//Declare grid size and block size
	int numblock = n / TILE + ((n % TILE) ? 1 : 0);
	dim3 dimGrid(numblock, numblock); //Dimensions of the grid in blocks
	dim3 dimBlock(TILE, TILE);// Dimensions of the block in threads

	//Allocate memory on device
	//hipMalloc((void**)&da, n * n * sizeof(float));
	hipMalloc((void**)&dl, n * n * sizeof(float));
	hipMalloc((void**)&du, n * n * sizeof(float));
	
	//Copy data to the device
	hipMemcpy(du, a[0], n * n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dl, L[0], n * n * sizeof(float), hipMemcpyHostToDevice);

	//Compute the LU decomposition for matrix a[n x n]
	//correct = ComputeLUDecomposition(a, L, n);

	//Do the matrix multiplication on the device (GPU)
	ComputeLUDecomposition << < dimGrid, dimBlock >> > (du, dl, n);

	// wait for the gpu to finish
	hipDeviceSynchronize();

	//Get results from the device
	hipMemcpy(L[0], dl, n * n * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(a[0], du, n * n * sizeof(float), hipMemcpyDeviceToHost);

	runtime = (clock() / (double)CLOCKS_PER_SEC) - runtime;


	//The eliminated matrix is as below:
	if (isPrintMatrix == 1)
	{
		cout << "\nLower Triangular Matrix:" << endl;
		PrintMatrix(L, n);
		cout << "\nUpper Triangular Matrix:" << endl;
		PrintMatrixU(a, n);
	}

	//print computing time
	cout << "\n\nLU Decomposition take: " << setiosflags(ios::fixed) << setprecision(8) << runtime << " seconds\n\n";
	cout << "Matrix size  = " << n << endl;

	//hipFree(da);
	hipFree(dl);
	hipFree(du);

	delete[] a[0];
	delete[] L[0];
	delete[] a;
	delete[] L;

	return 0;
}